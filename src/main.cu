#include "hip/hip_runtime.h"
/*
 * INF560
 *
 * Image Filtering Project
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>
#include <mpi.h>
#include "gif_lib.h"
#include "structs.h"
#include "vanilla_function.c"
#include "kernels.cu"

//#include "kernels.h"


/* Set this macro to 1 to enable debugging information */
#define SOBELF_DEBUG 0
#define USE_GPU 1
#define THREADS 256
/*
 * Load a GIF image from a file and return a
 * structure of type animated_gif.
 */
animated_gif * load_pixels( char * filename ) 
{
    GifFileType * g ;
    ColorMapObject * colmap ;
    int error ;
    int n_images ;
    int * width ;
    int * height ;
    pixel ** p ;
    int i ;
    animated_gif * image ;

    /* Open the GIF image (read mode) */
    g = DGifOpenFileName( filename, &error ) ;
    if ( g == NULL ) 
    {
        fprintf( stderr, "Error DGifOpenFileName %s\n", filename ) ;
        return NULL ;
    }

    /* Read the GIF image */
    error = DGifSlurp( g ) ;
    if ( error != GIF_OK )
    {
        fprintf( stderr, 
                "Error DGifSlurp: %d <%s>\n", error, GifErrorString(g->Error) ) ;
        return NULL ;
    }

    /* Grab the number of images and the size of each image */
    n_images = g->ImageCount ;

    width = (int *)malloc( n_images * sizeof( int ) ) ;
    if ( width == NULL )
    {
        fprintf( stderr, "Unable to allocate width of size %d\n",
                n_images ) ;
        return 0 ;
    }

    height = (int *)malloc( n_images * sizeof( int ) ) ;
    if ( height == NULL )
    {
        fprintf( stderr, "Unable to allocate height of size %d\n",
                n_images ) ;
        return 0 ;
    }

    /* Fill the width and height */
    for ( i = 0 ; i < n_images ; i++ ) 
    {
        width[i] = g->SavedImages[i].ImageDesc.Width ;
        height[i] = g->SavedImages[i].ImageDesc.Height ;

#if SOBELF_DEBUG
        printf( "Image %d: l:%d t:%d w:%d h:%d interlace:%d localCM:%p\n",
                i, 
                g->SavedImages[i].ImageDesc.Left,
                g->SavedImages[i].ImageDesc.Top,
                g->SavedImages[i].ImageDesc.Width,
                g->SavedImages[i].ImageDesc.Height,
                g->SavedImages[i].ImageDesc.Interlace,
                g->SavedImages[i].ImageDesc.ColorMap
                ) ;
#endif
    }


    /* Get the global colormap */
    colmap = g->SColorMap ;
    if ( colmap == NULL ) 
    {
        fprintf( stderr, "Error global colormap is NULL\n" ) ;
        return NULL ;
    }

#if SOBELF_DEBUG
    printf( "Global color map: count:%d bpp:%d sort:%d\n",
            g->SColorMap->ColorCount,
            g->SColorMap->BitsPerPixel,
            g->SColorMap->SortFlag
            ) ;
#endif

    /* Allocate the array of pixels to be returned */
    p = (pixel **)malloc( n_images * sizeof( pixel * ) ) ;
    if ( p == NULL )
    {
        fprintf( stderr, "Unable to allocate array of %d images\n",
                n_images ) ;
        return NULL ;
    }

    for ( i = 0 ; i < n_images ; i++ ) 
    {
        p[i] = (pixel *)malloc( width[i] * height[i] * sizeof( pixel ) ) ;
        if ( p[i] == NULL )
        {
        fprintf( stderr, "Unable to allocate %d-th array of %d pixels\n",
                i, width[i] * height[i] ) ;
        return NULL ;
        }
    }
    
    /* Fill pixels */

    /* For each image */
    for ( i = 0 ; i < n_images ; i++ )
    {
        int j ;

        /* Get the local colormap if needed */
        if ( g->SavedImages[i].ImageDesc.ColorMap )
        {

            /* TODO No support for local color map */
            fprintf( stderr, "Error: application does not support local colormap\n" ) ;
            return NULL ;

            colmap = g->SavedImages[i].ImageDesc.ColorMap ;
        }

        /* Traverse the image and fill pixels */
        for ( j = 0 ; j < width[i] * height[i] ; j++ ) 
        {
            int c ;

            c = g->SavedImages[i].RasterBits[j] ;

            p[i][j].r = colmap->Colors[c].Red ;
            p[i][j].g = colmap->Colors[c].Green ;
            p[i][j].b = colmap->Colors[c].Blue ;
        }
    }

    /* Allocate image info */
    image = (animated_gif *)malloc( sizeof(animated_gif) ) ;
    if ( image == NULL ) 
    {
        fprintf( stderr, "Unable to allocate memory for animated_gif\n" ) ;
        return NULL ;
    }

    /* Fill image fields */
    image->n_images = n_images ;
    image->width = width ;
    image->height = height ;
    image->p = p ;
    image->g = g ;

#if SOBELF_DEBUG
    printf( "-> GIF w/ %d image(s) with first image of size %d x %d\n",
            image->n_images, image->width[0], image->height[0] ) ;
#endif

    return image ;
}

int 
output_modified_read_gif( char * filename, GifFileType * g ) 
{
    GifFileType * g2 ;
    int error2 ;

#if SOBELF_DEBUG
    printf( "Starting output to file %s\n", filename ) ;
#endif

    g2 = EGifOpenFileName( filename, false, &error2 ) ;
    if ( g2 == NULL )
    {
        fprintf( stderr, "Error EGifOpenFileName %s\n",
                filename ) ;
        return 0 ;
    }

    g2->SWidth = g->SWidth ;
    g2->SHeight = g->SHeight ;
    g2->SColorResolution = g->SColorResolution ;
    g2->SBackGroundColor = g->SBackGroundColor ;
    g2->AspectByte = g->AspectByte ;
    g2->SColorMap = g->SColorMap ;
    g2->ImageCount = g->ImageCount ;
    g2->SavedImages = g->SavedImages ;
    g2->ExtensionBlockCount = g->ExtensionBlockCount ;
    g2->ExtensionBlocks = g->ExtensionBlocks ;

    error2 = EGifSpew( g2 ) ;
    if ( error2 != GIF_OK ) 
    {
        fprintf( stderr, "Error after writing g2: %d <%s>\n", 
                error2, GifErrorString(g2->Error) ) ;
        return 0 ;
    }

    return 1 ;
}


int
store_pixels( char * filename, animated_gif * image )
{
    int n_colors = 0 ;
    pixel ** p ;
    int i, j, k ;
    GifColorType * colormap ;

    /* Initialize the new_p set of colors */
    colormap = (GifColorType *)malloc( 256 * sizeof( GifColorType ) ) ;
    if ( colormap == NULL ) 
    {
        fprintf( stderr,
                "Unable to allocate 256 colors\n" ) ;
        return 0 ;
    }

    /* Everything is white by default */
    for ( i = 0 ; i < 256 ; i++ ) 
    {
        colormap[i].Red = 255 ;
        colormap[i].Green = 255 ;
        colormap[i].Blue = 255 ;
    }

    /* Change the background color and store it */
    int moy ;
    moy = (
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Red
            +
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Green
            +
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Blue
            )/3 ;
    if ( moy < 0 ) moy = 0 ;
    if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
    printf( "[DEBUG] Background color (%d,%d,%d) -> (%d,%d,%d)\n",
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Red,
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Green,
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Blue,
            moy, moy, moy ) ;
#endif

    colormap[0].Red = moy ;
    colormap[0].Green = moy ;
    colormap[0].Blue = moy ;

    image->g->SBackGroundColor = 0 ;

    n_colors++ ;

    /* Process extension blocks in main structure */
    for ( j = 0 ; j < image->g->ExtensionBlockCount ; j++ )
    {
        int f ;

        f = image->g->ExtensionBlocks[j].Function ;
        if ( f == GRAPHICS_EXT_FUNC_CODE )
        {
            int tr_color = image->g->ExtensionBlocks[j].Bytes[3] ;

            if ( tr_color >= 0 &&
                    tr_color < 255 )
            {

                int found = -1 ;

                moy = 
                    (
                     image->g->SColorMap->Colors[ tr_color ].Red
                     +
                     image->g->SColorMap->Colors[ tr_color ].Green
                     +
                     image->g->SColorMap->Colors[ tr_color ].Blue
                    ) / 3 ;
                if ( moy < 0 ) moy = 0 ;
                if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
                printf( "[DEBUG] Transparency color image %d (%d,%d,%d) -> (%d,%d,%d)\n",
                        i,
                        image->g->SColorMap->Colors[ tr_color ].Red,
                        image->g->SColorMap->Colors[ tr_color ].Green,
                        image->g->SColorMap->Colors[ tr_color ].Blue,
                        moy, moy, moy ) ;
#endif

                for ( k = 0 ; k < n_colors ; k++ )
                {
                    if ( 
                            moy == colormap[k].Red
                            &&
                            moy == colormap[k].Green
                            &&
                            moy == colormap[k].Blue
                       )
                    {
                        found = k ;
                    }
                }
                if ( found == -1  ) 
                {
                    if ( n_colors >= 256 ) 
                    {
                        fprintf( stderr, 
                                "Error: Found too many colors inside the image\n"
                               ) ;
                        return 0 ;
                    }

#if SOBELF_DEBUG
                    printf( "[DEBUG]\tnewa color %d\n",
                            n_colors ) ;
#endif

                    colormap[n_colors].Red = moy ;
                    colormap[n_colors].Green = moy ;
                    colormap[n_colors].Blue = moy ;


                    image->g->ExtensionBlocks[j].Bytes[3] = n_colors ;

                    n_colors++ ;
                } else
                {
#if SOBELF_DEBUG
                    printf( "[DEBUG]\tFound existing color %d\n",
                            found ) ;
#endif
                    image->g->ExtensionBlocks[j].Bytes[3] = found ;
                }
            }
        }
    }

    for ( i = 0 ; i < image->n_images ; i++ ) // for every image
    {
        for ( j = 0 ; j < image->g->SavedImages[i].ExtensionBlockCount ; j++ )
        {
            int f ;

            f = image->g->SavedImages[i].ExtensionBlocks[j].Function ;
            if ( f == GRAPHICS_EXT_FUNC_CODE )
            {
                int tr_color = image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] ;

                if ( tr_color >= 0 &&
                        tr_color < 255 )
                {

                    int found = -1 ;

                    moy = 
                        (
                         image->g->SColorMap->Colors[ tr_color ].Red
                         +
                         image->g->SColorMap->Colors[ tr_color ].Green
                         +
                         image->g->SColorMap->Colors[ tr_color ].Blue
                        ) / 3 ;
                    if ( moy < 0 ) moy = 0 ;
                    if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
                    printf( "[DEBUG] Transparency color image %d (%d,%d,%d) -> (%d,%d,%d)\n",
                            i,
                            image->g->SColorMap->Colors[ tr_color ].Red,
                            image->g->SColorMap->Colors[ tr_color ].Green,
                            image->g->SColorMap->Colors[ tr_color ].Blue,
                            moy, moy, moy ) ;
#endif

                    for ( k = 0 ; k < n_colors ; k++ )
                    {
                        if ( 
                                moy == colormap[k].Red
                                &&
                                moy == colormap[k].Green
                                &&
                                moy == colormap[k].Blue
                           )
                        {
                            found = k ;
                        }
                    }
                    if ( found == -1  ) 
                    {
                        if ( n_colors >= 256 ) 
                        {
                            fprintf( stderr, 
                                    "Error: Found too many colors inside the image\n"
                                   ) ;
                            return 0 ;
                        }

#if SOBELF_DEBUG
                        printf( "[DEBUG]\tnewa color %d\n",
                                n_colors ) ;
#endif

                        colormap[n_colors].Red = moy ;
                        colormap[n_colors].Green = moy ;
                        colormap[n_colors].Blue = moy ;


                        image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] = n_colors ;

                        n_colors++ ;
                    } else
                    {
#if SOBELF_DEBUG
                        printf( "[DEBUG]\tFound existing color %d\n",
                                found ) ;
#endif
                        image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] = found ;
                    }
                }
            }
        }
    }

#if SOBELF_DEBUG
    printf( "[DEBUG] Number of colors after background and transparency: %d\n",
            n_colors ) ;
#endif

    p = image->p ;

    /* Find the number of colors inside the image */
    for ( i = 0 ; i < image->n_images ; i++ )
    {

#if SOBELF_DEBUG
        printf( "OUTPUT: Processing image %d (total of %d images) -> %d x %d\n",
                i, image->n_images, image->width[i], image->height[i] ) ;
#endif

        for ( j = 0 ; j < image->width[i] * image->height[i] ; j++ ) 
        {
            int found = 0 ;
            for ( k = 0 ; k < n_colors ; k++ )
            {
                if ( p[i][j].r == colormap[k].Red &&
                        p[i][j].g == colormap[k].Green &&
                        p[i][j].b == colormap[k].Blue )
                {
                    found = 1 ;
                }
            }

            if ( found == 0 ) 
            {
                if ( n_colors >= 256 ) 
                {
                    fprintf( stderr, 
                            "Error: Found too many colors inside the image\n"
                           ) ;
                    return 0 ;
                }

#if SOBELF_DEBUG
                printf( "[DEBUG] Found new_p %d color (%d,%d,%d)\n",
                        n_colors, p[i][j].r, p[i][j].g, p[i][j].b ) ;
#endif

                colormap[n_colors].Red = p[i][j].r ;
                colormap[n_colors].Green = p[i][j].g ;
                colormap[n_colors].Blue = p[i][j].b ;
                n_colors++ ;
            }
        }
    }

#if SOBELF_DEBUG
    printf( "OUTPUT: found %d color(s)\n", n_colors ) ;
#endif


    /* Round up to a power of 2 */
    if ( n_colors != (1 << GifBitSize(n_colors) ) )
    {
        n_colors = (1 << GifBitSize(n_colors) ) ;
    }

#if SOBELF_DEBUG
    printf( "OUTPUT: Rounding up to %d color(s)\n", n_colors ) ;
#endif

    /* Change the color map inside the animated gif */
    ColorMapObject * cmo ;

    cmo = GifMakeMapObject( n_colors, colormap ) ;
    if ( cmo == NULL )
    {
        fprintf( stderr, "Error while creating a ColorMapObject w/ %d color(s)\n",
                n_colors ) ;
        return 0 ;
    }

    image->g->SColorMap = cmo ;

    /* Update the raster bits according to color map */
    for ( i = 0 ; i < image->n_images ; i++ )
    {
        for ( j = 0 ; j < image->width[i] * image->height[i] ; j++ ) 
        {
            int found_index = -1 ;
            for ( k = 0 ; k < n_colors ; k++ ) 
            {
                if ( p[i][j].r == image->g->SColorMap->Colors[k].Red &&
                        p[i][j].g == image->g->SColorMap->Colors[k].Green &&
                        p[i][j].b == image->g->SColorMap->Colors[k].Blue )
                {
                    found_index = k ;
                }
            }

            if ( found_index == -1 ) 
            {
                fprintf( stderr,
                        "Error: Unable to find a pixel in the color map\n" ) ;
                return 0 ;
            }

            image->g->SavedImages[i].RasterBits[j] = found_index ;
        }
    }


    /* Write the final image */
    if ( !output_modified_read_gif( filename, image->g ) ) { return 0 ; }

    return 1 ;
}


#define CONV(l,c,nb_c) \
    (l)*(nb_c)+(c)
void test (animated_gif * image)
{
    
}

/*
 * Main entry point
 */
int 
main( int argc, char ** argv )
{
   MPI_Init(NULL,NULL);
    
    char * input_filename ; 
    char * output_filename ;
    char * file_to_save;
    char * N;
    animated_gif * image ;
    struct timeval t1, t2;
    double loading_time,subgroup_time,sobel_time,gathering_time,export_time,full_time;
    double duration ;
    int rank,size,chunk_size,remainder,nb_threads;
    int true_chunk_size;
    animated_gif* subgroup;
    
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    int sendcounts;
    int pos_to_affect;
    int on_gpu;
    nb_threads  = THREADS;
    on_gpu = USE_GPU;
    file_to_save = "collecting_cuda.csv";
    // we need to define a new_p MPI_Datatype MPI_PIXEL
    MPI_Datatype MPI_PIXEL;
    MPI_Type_contiguous(3, MPI_INT, &MPI_PIXEL);
    MPI_Type_commit(&MPI_PIXEL);
    //printf("%d %d \n",rank,size);
    /* Check command-line arguments */
    if ( argc < 4 )
    {
        fprintf( stderr, "Usage: %s input.gif output.gif \n", argv[0] ) ;
        return 1 ;
    }
     input_filename = argv[1] ;
    output_filename = argv[2] ;
    N = argv[3];
    int opt;
    while ((opt = getopt(argc, argv, "t:g:f:")) != -1) {
        
        switch (opt) {
        case 't':
            nb_threads = atoi(optarg);
            break;
        case 'g':
            on_gpu = atoi(optarg);
            break;
        case 'f':
            file_to_save = optarg;
            break;
        }
    }


   

    /* IMPORT Timer start */
    gettimeofday(&t1, NULL);

    /* Load file and store the pixels in array */
    //every process load the image
    image = load_pixels( input_filename ) ;
    // On traite le cas d'un nombre d'images non divisible par size
    gettimeofday(&t2, NULL);

    loading_time = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);
    on_gpu =  on_gpu && test_gpu_available(image->width[0],image->height[0]);
    int images_restantes;
    if ( image == NULL ) { return 1 ; }
    if(on_gpu)
    {
       
        // we have a new_p time : the subgroup allocation time
        gettimeofday(&t1, NULL);
        int chunk_size_0;
        chunk_size_0 = 2 * image->n_images / (size +1);
        images_restantes = image->n_images - 2 * image->n_images / (size +1);
        if(rank == 0)
        {
            
            chunk_size = chunk_size_0;
            true_chunk_size = chunk_size;
            pos_to_affect = 0;
        }
        else
        {
            chunk_size = images_restantes / (size-1);
            remainder = images_restantes % (size-1);
        //Each one will determine how many images it will have to filters, and which ones
            int rank_b;
            rank_b =rank-1;
            pos_to_affect = 2 * image->n_images / (size +1) + chunk_size * rank_b + (rank_b < remainder ? rank_b : remainder);
            true_chunk_size = chunk_size + (rank_b < remainder ? 1 : 0);
        }
    }
    else
    {
        chunk_size = image->n_images / size;
        remainder = image->n_images % size;
    // Each one will determine how many images it will have to filters, and which ones
        pos_to_affect = chunk_size * rank + (rank < remainder ? rank : remainder);
        true_chunk_size = chunk_size + (rank < remainder ? 1 : 0);
    }
    //Allocation
    subgroup = (animated_gif*)malloc(sizeof(animated_gif));
    subgroup->width = (int*)malloc(true_chunk_size * sizeof(int));
    subgroup->height = (int*)malloc(true_chunk_size * sizeof(int));
    subgroup->n_images = true_chunk_size;
    subgroup->p = (pixel**) malloc(true_chunk_size  * sizeof(pixel *));

     
    // We can't use scatterv with image->p because there are memory problems : les pointeurs sont différents dans chaque process à cause de l'adressage virtuel
   
    
    /* IMPORT Timer stop */
    // Each one treat the good images
    int i;
    for(i = 0 ; i<true_chunk_size;i++)
    {
        subgroup->height[i] = image->height[pos_to_affect +i];
        subgroup->width[i] = image->width[pos_to_affect +i];
        subgroup->p[i] = image->p[pos_to_affect +i];
    }

    gettimeofday(&t2,NULL);
    subgroup_time = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6); 
    
    
    //printf("Nombre maximum de blocs pour %d threads: %d\n", total_threads, max_blocks);

    
    
    /* FILTER Timer start */
    //printf("before time t1\n");
    gettimeofday(&t1, NULL);
    if(rank == 0 && on_gpu )
    {
        on_gpu = 1;
        apply_gray_filter_cuda(subgroup,nb_threads) ;
   
        apply_blur_filter_cuda(subgroup,5,20,nb_threads);

        apply_sobel_filter_cuda(subgroup,nb_threads) ;

    }
    else
    {   
        on_gpu = 0;
        apply_gray_filter_v(subgroup) ;

        apply_blur_filter_v(subgroup,5,20);
    
        apply_sobel_filter_v(subgroup) ;
        
    }
    
    gettimeofday(&t2, NULL);
    
    sobel_time = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);


    //printf("%lf \n",export_time);

    /* EXPORT Timer start */
    

    /* Store file from array of pixels to GIF file */
    
     if(rank == 0)
    {
        
    // rank 0 doit recevoir les données de tout le monde ça c'est galère
    //gathering Time
        int chunk_size_b,remainder_b;
        gettimeofday(&t1,NULL);
        int * pos_current_rank;
        pos_current_rank = (int*) malloc(size * sizeof(int));
        for(i = 0; i < size; i ++)
        {
            pos_current_rank[i] =0;
        }
        // On va recevoir n_images communications 
        // attention comme on ne sait pas quelle est la taille de l'image recu cela ne marche que si toutes les images on a la même taille

        MPI_Status status;
        if(on_gpu)
        {
            for(i = 0; i < image->n_images - true_chunk_size; i++)
            {
                MPI_Status status;
                
                //printf("We are waiting for someting \n");
                pixel* tmp = (pixel*)malloc(image->width[0] * image->height[0] * sizeof(pixel));
                MPI_Recv(tmp,image->width[0] * image->height[0],MPI_PIXEL,MPI_ANY_SOURCE,0,MPI_COMM_WORLD,&status);
                //printf("We received something from %d \n",status.MPI_SOURCE);
                chunk_size_b = images_restantes / (size-1);
                remainder_b = images_restantes % (size-1);
            //Each one will determine how many images it will have to filters, and which ones
                int rank_b;
                rank_b = status.MPI_SOURCE-1;
                pos_to_affect = 2 * image->n_images / (size +1) + chunk_size_b * rank_b + (rank_b < remainder_b ? rank_b : remainder_b);
                image->p[pos_to_affect +pos_current_rank[status.MPI_SOURCE]] = tmp;
                pos_current_rank[status.MPI_SOURCE]++;
            }
        }
        else
        {
            for(i = 0; i < image->n_images - true_chunk_size; i++)
            {
                MPI_Status status;
                //printf("We are waiting for someting \n");
                pixel* tmp = (pixel*)  malloc(image->width[0] * image->height[0] * sizeof(pixel));
                
                MPI_Recv(tmp,image->width[0] * image->height[0],MPI_PIXEL,MPI_ANY_SOURCE,0,MPI_COMM_WORLD,&status);
                //printf("We received something from %d \n",status.MPI_SOURCE);
                image->p[status.MPI_SOURCE * chunk_size + (status.MPI_SOURCE < remainder ? i : remainder) +pos_current_rank[status.MPI_SOURCE]] = tmp;
                pos_current_rank[status.MPI_SOURCE]++;
            }
        }
        gettimeofday(&t2,NULL);
        gathering_time =(t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);
        
        gettimeofday(&t1, NULL);
        
        

    }
    else
    {
        int j;
        for(j = 0;j< true_chunk_size;j++)
        {
            MPI_Send(subgroup->p[j],subgroup->width[j] * subgroup->height[j],MPI_PIXEL,0,0,MPI_COMM_WORLD);
        }
    }
    /* EXPORT Timer stop */
    

    //freeing memory 
    MPI_Barrier(MPI_COMM_WORLD);
    if(rank ==0)
    {
    if ( !store_pixels( output_filename, image ) ) { return 1 ; }
        printf("file done \n");
        gettimeofday(&t2, NULL);
        
        export_time = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);
         FILE *f = fopen(file_to_save,"a");
        full_time = loading_time  + sobel_time + export_time;
        fprintf(f,"%s;%d;%d;%d;%d;%d;%lf;%lf;%lf;%lf \n",input_filename,image->n_images,image->width[0] * image->height[0],on_gpu,nb_threads,size,loading_time,sobel_time,export_time,full_time);
        fclose(f);
    }
    
    free(subgroup->height);
    free(subgroup->width);
    
    free(subgroup->p);
        //printf("Done for someone \n");
   
    
    MPI_Finalize();
    return 0 ;
}

