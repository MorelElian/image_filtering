#include "hip/hip_runtime.h"
#define MEMORY_PER_THREAD 10
#define N_IMAGES_PER_ROUND 2
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "gif_lib.h"
__global__ void apply_gray_filter_kernel(pixel * p, int width, int height)
{
    int j;
    j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= width * height) return;

    int moy;
    moy = (p[j].r + p[j].g + p[j].b) / 3;
    if (moy < 0) moy = 0;
    if (moy > 255) moy = 255;

    p[j].r = moy;
    p[j].g = moy;
    p[j].b = moy;
}
extern "C" void apply_gray_filter_cuda(animated_gif *image,int nb_threads)
{
    pixel **p;
    p = image->p;
    int x;
    int n_images = image->n_images;
    int width;
    int height;
    
    int nb_blocks;
    size_t size_pixels;
    int n_pixels;
    for (int i = 0; i < n_images; i++) {
        pixel* d_p;
        n_pixels = image->width[i] * image->height[i];
        size_pixels = n_pixels * sizeof(pixel);
        nb_threads = 256;
        nb_blocks = (size_pixels / nb_threads)+1;
        width = image->width[i];
        height = image->height[i];

        hipMalloc(&d_p, size_pixels);
        hipMemcpy(d_p, p[i], size_pixels, hipMemcpyHostToDevice);
        
        apply_gray_filter_kernel<<<nb_blocks, nb_threads>>>(d_p, width, height);
        hipDeviceSynchronize();
        
        hipMemcpy(p[i], d_p, size_pixels, hipMemcpyDeviceToHost);
        hipFree(d_p);
        
    }
}
__global__ void blur_image_kernel(pixel* p_cuda, pixel* new_p,int width,int height,int size)
{
    //indice du point
    
    int i  = blockIdx.x * blockDim.x +threadIdx.x;
    int j = i / width; // peut valoir en 0 et height-1 equivalent de j dans la version originale
    int k = i % width; // peut valoir en 0 et width-1 equivalent de k
    if(i >= width * height) return;
    else
    {
        //There are 3 cases : top/middle/bottom of the image
        if(j >= size && j < height/10 -size && k>=size && k <width-size)
        {
            int stencil_j, stencil_k ;
            int t_r = 0 ;
            int t_g = 0 ;
            int t_b = 0 ;
            for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
                    {
                        for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
                        {
                            t_r += p_cuda[(j+stencil_j) * width + k].r ;
                            t_g += p_cuda[(j+stencil_j)*width + k+stencil_k].g ;
                            t_b += p_cuda[(j+stencil_j)*width + k+stencil_k].b ;
                        }
                    }
            new_p[j*width +k].r = t_r / ( (2*size+1)*(2*size+1) ) ;
            new_p[j*width +k].g = t_g / ( (2*size+1)*(2*size+1) ) ;
            new_p[j*width +k].b = t_b / ( (2*size+1)*(2*size+1) ) ;
        }
        else if(j>=height/10 -size && j < height * 0.9+size && k >= size && k < width - size)
        {
            new_p[width*j + k].r = p_cuda[width*j + k].r ; 
            new_p[width*j + k].g = p_cuda[width*j + k].g ; 
            new_p[width*j + k].b = p_cuda[width*j + k].b ;
        }
        else if(j>=0.9*height + size && j < height-size && k>=size && k < width-size)
        {
            int stencil_j, stencil_k ;
            int t_r = 0 ;
            int t_g = 0 ;
            int t_b = 0 ;
            for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
                    {
                        for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
                        {
                            t_r += p_cuda[(j+stencil_j)*width + k+stencil_k].r ;
                            t_g += p_cuda[(j+stencil_j)*width + k+stencil_k].g ;
                            t_b += p_cuda[(j+stencil_j)*width + k+stencil_k].b ;
                        }
                    }
            new_p[j*width +k].r = t_r / ( (2*size+1)*(2*size+1) ) ;
            new_p[j*width +k].g = t_g / ( (2*size+1)*(2*size+1) ) ;
            new_p[j*width +k].b = t_b / ( (2*size+1)*(2*size+1) ) ;
        }
        else
        {
            new_p[width*j + k].r = p_cuda[width*j + k].r ; 
            new_p[width*j + k].g = p_cuda[width*j + k].g ; 
            new_p[width*j + k].b = p_cuda[width*j + k].b ;
        }
    }
}
__global__ void test_blur_image_kernel(pixel * p_cuda, pixel* new_p,int* end, int width, int height, int threshold)
{
    int i  = blockIdx.x * blockDim.x +threadIdx.x;
    int j = i / width; // peut valoir en 0 et height-1 equivalent de j dans la version originale
    int k = i %width; // peut valoir en 0 et width-1 equivalent de k
    
    if(i >= width * height) return;
    float diff_r ;
    float diff_g ;
    float diff_b ;
    if(j >= 0 && k >=0)
    {
        diff_r = (new_p[j*width +k].r - p_cuda[j*width +k].r) ;
        diff_g = (new_p[j*width +k].g - p_cuda[j*width +k].g) ;
        diff_b = (new_p[j*width +k].b - p_cuda[j*width +k].b);
       
        if ( diff_r > threshold || -diff_r > threshold 
                ||
                    diff_g > threshold || -diff_g > threshold
                    ||
                    diff_b > threshold || -diff_b > threshold
            ) {
            
            *(end) = 0 ;
        }
    }
    p_cuda[width*j + k].r = new_p[width*j + k].r ;
    p_cuda[width*j + k].g = new_p[width*j + k].g ;
    p_cuda[width*j + k].b = new_p[width*j + k].b ;
}
extern "C" void
apply_blur_filter_cuda( animated_gif * image, int size, int threshold ,int nb_threads)
{
   
    int width, height,nb_blocks ;
    int x;
    x = 1;
    int end;
    end = 0;
    
    size_t size_pixels;
    pixel ** p ;
    
    int i;
    
    
    /* Get the pixels of all images */
    p = image->p ;


    /* Process all images */
    for ( i = 0 ; i < image->n_images ; i++ )
    {
        
        width = image->width[i] ;
        height = image->height[i] ;
        
        
        size_pixels = width * height * sizeof(pixel);
        nb_blocks = width*height / nb_threads +1;
        
        /* Allocate array of newa pixels */
        pixel * new_p ;
        pixel* p_cuda;
        hipMalloc(&new_p,size_pixels);
        hipMalloc(&p_cuda,size_pixels);
        hipMemcpy(p_cuda, p[i], size_pixels, hipMemcpyHostToDevice);
        do
        {
            hipDeviceSynchronize();
            blur_image_kernel<<<nb_blocks,nb_threads>>>(p_cuda,new_p,width,height,size);
            hipDeviceSynchronize();
            end = 1;
          
            int *end_cuda;
            hipMalloc(&end_cuda,sizeof(int));
            hipMemcpy(end_cuda,&end,sizeof(int),hipMemcpyHostToDevice);
            test_blur_image_kernel<<<nb_blocks,nb_threads>>>(p_cuda,new_p,end_cuda,width,height,threshold);
            hipDeviceSynchronize();
            hipMemcpy(&end,end_cuda,sizeof(int),hipMemcpyDeviceToHost);
            hipFree(end_cuda);
            
        } while (threshold >0 && !end);
        hipMemcpy(p[i],p_cuda,size_pixels,hipMemcpyDeviceToHost);
        

        hipFree(p_cuda);
        hipFree(new_p);
    }
    
}
__global__ void apply_sobel_filter_kernel(pixel* p_cuda,pixel* sobel, int width,int height)
{
    int i,j,k;
    i = blockIdx.x* blockDim.x + threadIdx.x;
    j = i /width;
    k = i%width;
    
    //int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
   

    float deltaX_blue ;
    float deltaY_blue ;
    float val_blue;

    //pixel_blue_no = p_cuda[(j-1)*width + k-1].b ;
    //pixel_blue_n  = p_cuda[width * (j-1) +k  ].b ;
    //pixel_blue_ne = p_cuda[width * (j-1) +k+1].b ;
    
    
    deltaX_blue = -p_cuda[(j-1)*width + k-1].b  + p_cuda[width * (j-1) +k+1].b - 2*p_cuda[width * (j  ) +k-1].b  + 2*p_cuda[width * (j  ) +k+1].b - p_cuda[width * (j+1) +k-1].b + p_cuda[width * (j+1) +k+1].b ;          

    deltaY_blue = p_cuda[width * (j+1) +k+1].b  + 2*p_cuda[width * (j+1) +k  ].b  + p_cuda[width * (j+1) +k-1].b - p_cuda[width * (j-1) +k+1].b - 2*p_cuda[width * (j-1) +k  ].b -p_cuda[(j-1)*width + k-1].b ;

    val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue)/4; 
   
    //printf("%f \n",val_blue);
    if ( val_blue > 50 ) 
    {
        //printf("aaa\n");
        sobel[width*j + k].r = 254 ;
        sobel[width*j + k].g = 254 ;
        sobel[width*j + k].b = 254 ;
    } else
    {

        sobel[width*j + k].r = 0 ;
        sobel[width*j + k].g = 0 ;
        sobel[width*j + k].b = 0 ;
    }
    
}
extern "C" void apply_sobel_filter_cuda(animated_gif* image,int nb_threads)
{
    int i,width,height;

    int nb_blocks;

    pixel**p;
     p = image->p ;
    
    for(i = 0 ;i < image->n_images; i++)
    {
        
        width = image->width[i] ;
        height = image->height[i] ;
        
        nb_blocks = width*height /nb_threads +1;
        pixel* sobel;
        pixel* p_cuda;
        hipMalloc(&sobel,width*height*sizeof(pixel));
        hipMalloc(&p_cuda,width*height*sizeof(pixel));
        
        hipMemcpy(p_cuda,p[i],width*height*sizeof(pixel),hipMemcpyHostToDevice);
        apply_sobel_filter_kernel<<<nb_blocks,nb_threads>>>(p_cuda,sobel,width,height);
        hipDeviceSynchronize();
        
    
        hipFree(p_cuda);
        hipMemcpy(p[i],sobel,width*height*sizeof(pixel),hipMemcpyDeviceToHost);
        hipFree(sobel);
        
    }
}
bool test_gpu_available(int width,int height)
// This function tests two things : first is there a gpu available on the node second is there enough memory one the gpu to do the computation
{
     int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess || deviceCount == 0) {
        return false;
    }
    else
    {
        int device ; // ID du périphérique CUDA à utiliser (dans ce cas, le premier périphérique)
        device = 0;
        size_t free_byte, total_byte;
        hipSetDevice(device);
        hipMemGetInfo(&free_byte, &total_byte);
        double free_db = (double)free_byte;
        free_db /= 1024.0;
        free_db /= 1024.0;
        double needed_memory;
        needed_memory = width/1024.0    * height /1024.0 *sizeof(pixel) *MEMORY_PER_THREAD * N_IMAGES_PER_ROUND  *sizeof(int);
        if(needed_memory > free_db * 0.8)
        {
            return false;
        }
        return true;
     
    }
}